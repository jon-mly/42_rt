#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_vector		cone_norml(t_object ray, t_object cone)
{
	t_vector	distance;
	float		normal_dist;
	t_point		normal_point;
	t_point		normal_point_2;
	t_vector	normal;

	distance = vector_pts(cone.center, ray.intersectiion);
	distance = rotate_vec_angles(cone, distance, 0);
	normal_dist = (cosf(cone.angle) + tan(cone.angle) * sinf(cone.angle)) *
		vec_norm(distance);
	normal_point = (t_point){0, 0, normal_dist};
	normal_point_2 = (t_point){0, 0, -normal_dist};
	if (pts_norm(normal_point, distance) > pts_norm(normal_point_2,
				distance))
		normal_point = normal_point_2;
	normal = vector_pts(normal_point, distance);
	if (dot_prod(normalize_vec(normal), rotate_vec_angles(cone, ray.direction, 0)) > 0)
		normal = vector_pts(distance, normal_point);
	normal = rotate_vec_angles(cone, normal, 1);
	return (normalize_vec(normal));
}

__device__ int			revert_sphere_normal(t_object ray, t_object sphere)
{
	t_vector	distance_vector;

	distance_vector = vector_pts(ray.origin, sphere.center);
	return (vec_norm(distance_vector) < sphere.radius);
}
