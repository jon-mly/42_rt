#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_vector			shape_norml(t_object ray, t_object object)
{
	if (object.typpe == SPHERE)
		return (sphere_norml(ray, object));
	else if (object.typpe == PLANE || object.typpe == DISC || object.typpe == RECTANGLE
		|| object.typpe == TRIANGLE || object.typpe == PARALLELOGRAM)
		return (plane_norml(ray, object));
	else if (object.typpe == CYLINDER)
		return (cylinder_norml(ray, object));
	else if (object.typpe == HYPERBOLOID)
		return (hyperboloid_norml(ray, object));
	else
		return (cone_norml(ray, object));
}

__device__ t_vector			hyperboloid_norml(t_object ray, t_object hyperboloid)
{
	t_vector	normal;
	t_vector	distance;

	distance = vector_pts(hyperboloid.center, ray.intersectiion);
	distance = rotate_vec_angles(hyperboloid, distance, 0);
	normal = (t_vector){
		distance.x,
		distance.y,
		-pow(hyperboloid.rho / hyperboloid.sigma, 2) * distance.z
	};
	normal = rotate_vec_angles(hyperboloid, normal, 1);
	return (normalize_vec(normal));
}

__device__ t_vector			sphere_norml(t_object ray, t_object sphere)
{
	t_vector	normal;

	if (revert_sphere_normal(ray, sphere))
		normal = vector_pts(ray.intersectiion, sphere.center);
	else
		normal = vector_pts(sphere.center, ray.intersectiion);
	return (normalize_vec(normal));
}

__device__ t_vector			plane_norml(t_object ray, t_object plane)
{
	t_vector	normal;

	if (dot_prod(ray.direction, plane.normal) <= 0)
		return (normalize_vec(plane.normal));
	normal.x = -(plane.normal.x);
	normal.y = -(plane.normal.y);
	normal.z = -(plane.normal.z);
	return (normalize_vec(normal));
}

__device__ t_vector			cylinder_norml(t_object ray, t_object cylinder)
{
	t_vector	distance;
	t_point		normal_point;
	t_vector	normal;

	distance = vector_pts(cylinder.point, ray.intersectiion);
	distance = rotate_vec_angles(cylinder, distance, 0);
	normal_point = (t_point){0, 0, distance.z};
	normal = vector_pts(normal_point, distance);
	if (dot_prod(normalize_vec(normal), rotate_vec_angles(cylinder, ray.direction, 0)) > 0)
		normal = vector_pts(distance, normal_point);
	normal = rotate_vec_angles(cylinder, normal, 1);
	return (normalize_vec(normal));
}
