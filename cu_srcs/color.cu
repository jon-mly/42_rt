#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_color		average_color(t_color c1, t_color c2)
{
	t_color		final;

	final.r = (c1.r + c2.r) / 2;
	final.g = (c1.g + c2.g) / 2;
	final.b = (c1.b + c2.b) / 2;
	final.a = (c1.a + c2.a) / 2;
	return (final);
}

__device__ int			color_to_int(t_color color)
{
	int 	r;
	int 	g;
	int		b;
	int 	a;

	r = (int)color.r;
	g = (int)color.g;
	b = (int)color.b;
	a = (int)color.a;
	return (a << 24 | r << 16 | g << 8 | b);
}

__device__ t_color		add_color(t_color base, t_color overlay)
{
	t_color 	final;

	final.r = maximize_color_value(base.r + overlay.r);
	final.g = maximize_color_value(base.g + overlay.g);
	final.b = maximize_color_value(base.b + overlay.b);
	final.a = maximize_color_value(base.a + overlay.a);
	return (final);
}

__device__ unsigned char	maximize_color_value(int color_value)
{
	return ((unsigned char)(fmaxf(fminf((float)color_value, (float)255), 0)));
}

__device__ t_color		fade_color(t_color color, float multiplier)
{
	color.r = (unsigned char)((float)color.r * multiplier);
	color.g = (unsigned char)((float)color.g * multiplier);
	color.b = (unsigned char)((float)color.b * multiplier);
	color.a = (unsigned char)((float)color.a * multiplier);
	return (color);
}

__device__ float		pts_norm(t_point p1, t_point p2)
{
	float		distance;

	distance = sqrt((float)(pow((float)(p2.x - p1.x), (float)2) + pow((float)(p2.y - p1.y), (float)2) + pow((float)(p2.z - p1.z), (float)2)));
	return (distance);
}

__device__ int			colors_are_equals(t_color c1, t_color c2)
{
	return (c1.r == c2.r
		&& c1.g == c2.g
		&& c1.b == c2.b
		&& c1.a == c2.a);
}
