#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_color			specular_light_for_intersection(t_object light_ray, t_object ray, t_object object, t_light light)
{
	float		distance;
	float		intensity;
	t_vector	incident;
	t_vector	reflected;
	t_color		specular;

	if (light.typpe == PROJECTOR && dot_prod(light_ray.direction, light.direction) < cosf(light.angle))
		return (BLACK);
	incident = scale_vector(light_ray.direction, -1);
	distance = (light.typpe != AMBIANT)
		? pts_norm(ray.intersectiion, light_ray.origin) * (100.0 / light.power)
		: 100.0 * (100.0 / light.power);
		distance = 1;
	reflected = reflected_vector(incident, shape_norml(ray, object));
	intensity = powf(fmaxf(dot_prod(reflected, ray.direction), 0), (int)(object.brillance * 100)) * powf(object.brillance, 2);
	intensity *= (1 - object.transparency);
	if (light.typpe == PROJECTOR)
		intensity *= (1 / (1 - cosf(light.angle))) * dot_prod(light.direction, light_ray.direction) - (cosf(light.angle) / (1 - cosf(light.angle)));
	specular.r = projector_color_coord(intensity, distance, object.color.r, light_ray.color.r);
	specular.g = projector_color_coord(intensity, distance, object.color.g, light_ray.color.g);
	specular.b = projector_color_coord(intensity, distance, object.color.b, light_ray.color.b);
	specular.a = 0;
	return (specular);
}

__device__ t_color			ambiant_light_for_intersection(t_object light_ray, t_object ray, t_object object, t_light light)
{
	t_color		color;
	float		distance;
	float		cosinus;

	cosinus = dot_prod(scale_vector(light_ray.direction, -1), shape_norml(ray, object)) * object.diffuse * (1 - object.transparency);
	if (cosinus <= 0)
		return (BLACK);
	distance = 100.0 * (100.0 / light.power);
	color.r = omni_color_coord(-cosinus, distance, object.color.r, light_ray.color.r);
	color.g = omni_color_coord(-cosinus, distance, object.color.g, light_ray.color.g);
	color.b = omni_color_coord(-cosinus, distance, object.color.b, light_ray.color.b);
	color.a = 0;
	return (color);
}

__device__ t_color			projector_light_for_intersection(t_object light_ray, t_object ray, t_object object, t_light light)
{
	t_vector	normal;
	float		cosinus;
	float		distance;
	float 		intensity;
	t_color		color;

	if (dot_prod(light.direction, light_ray.direction) < cosf(light.angle))
		return (BLACK);
	light_ray.intersectiion = ray.intersectiion;
	normal = shape_norml(ray, object);
	distance = pts_norm(ray.intersectiion, light_ray.origin) * (100.0 / light.power);
	cosinus = dot_prod(light.direction, light_ray.direction);
	intensity = (1 / (1 - cosf(light.angle))) * cosinus - (cosf(light.angle) / (1 - cosf(light.angle)));
	intensity *= object.diffuse * (1 - object.transparency);
	cosinus = dot_prod(light_ray.direction, normal);
	if (cosinus >= 0)
		return (BLACK);
	intensity = -cosinus * intensity;
	color.r = projector_color_coord(intensity, distance, object.color.r, light_ray.color.r) * object.diffuse;
	color.g = projector_color_coord(intensity, distance, object.color.g, light_ray.color.g) * object.diffuse;
	color.b = projector_color_coord(intensity, distance, object.color.b, light_ray.color.b) * object.diffuse;
	color.a = 0;
	return (color);
}

__device__ t_color			omni_light_for_intersection(t_object light_ray, t_object ray, t_object object, t_light light)
{
	t_vector	normal;
	float		cosinus;
	float		distance;
	t_color		color;

	light_ray.intersectiion = ray.intersectiion;
	normal = shape_norml(ray, object);
	cosinus = dot_prod(light_ray.direction, normal) * object.diffuse * (1 - object.transparency);
	if (cosinus >= 0)
		return (BLACK);
	distance = pts_norm(ray.intersectiion, light_ray.origin) * (100.0 / light.power);
	color.r = omni_color_coord(cosinus, distance, object.color.r, light_ray.color.r) * object.diffuse;
	color.g = omni_color_coord(cosinus, distance, object.color.g, light_ray.color.g) * object.diffuse;
	color.b = omni_color_coord(cosinus, distance, object.color.b, light_ray.color.b) * object.diffuse;
	color.a = 0;
	return (color);
}

__device__ int			projector_color_coord(float intensity, float distance, int obj_color, int light_color)
{
	float		color_value;
	float		distance_factor;
	float		k;

	distance_factor = 0.02 * pow((float)(distance / 1.3), (float)2) + 1;
	k = intensity / distance_factor;
	color_value = ((float)obj_color + (float)light_color) * k;
	color_value = maximize_color_value((int)color_value);
	return ((int)color_value);
}
