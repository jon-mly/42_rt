#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_object			intersect_obj(t_object ray, t_object object)
{
	if (object.typpe == SPHERE)
		ray = sphere_intersect(ray, object);
	else if (object.typpe == PLANE)
		ray = plane_intersect(ray, object);
	else if (object.typpe == CYLINDER)
		ray = (object.finite) 
			? finite_cylinder_intersection(ray, object) 
			: cylinder_intersect(ray, object);
	else if (object.typpe == CONE)
		ray = (object.finite)
			? finite_cone_intersection(ray, object)
			: cone_intersect(ray, object);
	else if (object.typpe == DISC)
		ray = disc_intersection(ray, object);
	else if (object.typpe == RECTANGLE)
		ray = rectangle_intersection(ray, object);
	else if (object.typpe == TRIANGLE)
		ray = triangle_intersection(ray, object);
	else if (object.typpe == PARALLELOGRAM)
		ray = parallelogram_intersection(ray, object);
	else if (object.typpe == HYPERBOLOID)
		ray = hyperboloid_intersection(ray, object);
	if (ray.intersect)
		ray.intersectiion = point_from_vec(ray.origin, ray.direction, ray.norm);
	return (ray);
}

__device__ t_object		disc_intersection(t_object ray, t_object disc)
{
	ray = plane_intersect(ray, disc);
	if (!ray.intersect)
		return (ray);
	ray.intersectiion = point_from_vec(ray.origin, ray.direction, ray.norm);
	ray.intersect = vec_norm(vector_pts(ray.intersectiion, disc.point)) < disc.radius;
	return (ray);
}

__device__ t_object		rectangle_intersection(t_object ray, t_object rectangle)
{
	t_vector	intersection_dist;

	ray = plane_intersect(ray, rectangle);
	if (!ray.intersect)
		return (ray);
	ray.intersectiion = point_from_vec(ray.origin, ray.direction, ray.norm);
	intersection_dist = vector_pts(rectangle.point, ray.intersectiion);
	intersection_dist = rotate_vec_angles(rectangle, intersection_dist, 0);
	ray.intersect = (intersection_dist.x >= 0 && intersection_dist.z >= 0
		&& intersection_dist.x < rectangle.width && intersection_dist.z < rectangle.height);
	return (ray);
}

__device__ t_object		triangle_intersection(t_object ray, t_object triangle)
{
	t_vector	distance;
	float		a;
	float		b;
	float		k;

	distance = vector_pts(triangle.point, ray.origin);
	k = -dot_prod(cross_prod(triangle.second_vect, triangle.first_vect), ray.direction);
	if (k == 0)
		return (ray);
	a = -dot_prod(cross_prod(distance, triangle.first_vect), ray.direction) / k;
	b = -dot_prod(cross_prod(triangle.second_vect, distance), ray.direction) / k;
	ray.intersect = (a >= 0 && b >= 0 && a + b < 1);
	ray.norm = dot_prod(cross_prod(triangle.second_vect, triangle.first_vect), distance) / k;
	return (ray);
}

__device__ t_object		parallelogram_intersection(t_object ray, t_object parallelogram)
{
	t_vector	distance;
	float		a;
	float		b;
	float		k;

	distance = vector_pts(parallelogram.point, ray.origin);
	k = -dot_prod(cross_prod(parallelogram.second_vect, parallelogram.first_vect), ray.direction);
	if (k == 0)
		return (ray);
	a = -dot_prod(cross_prod(distance, parallelogram.first_vect), ray.direction) / k;
	b = -dot_prod(cross_prod(parallelogram.second_vect, distance), ray.direction) / k;
	ray.intersect = (a >= 0 && b >= 0 && a < 1 && b < 1);
	ray.norm = dot_prod(cross_prod(parallelogram.second_vect, parallelogram.first_vect), distance) / k;
	return (ray);
}
