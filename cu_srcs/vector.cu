#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_vector		vector_pts(t_point p1, t_point p2)
{
	t_vector	vec;

	vec.x = p2.x - p1.x;
	vec.y = p2.y - p1.y;
	vec.z = p2.z - p1.z;
	return (vec);
}

__device__ t_vector		normalize_vec(t_vector vec)
{
	float		norm;

	norm = vec_norm(vec);
	vec.x /= norm;
	vec.y /= norm;
	vec.z /= norm;
	return (vec);
}

__device__ t_vector		point_from_vec(t_point origin, t_vector direction, float norm)
{
	t_point		point;

	point.x = origin.x + direction.x * norm;
	point.y = origin.y + direction.y * norm;
	point.z = origin.z + direction.z * norm;
	return (point);
}

__device__ t_vector		reflected_vector(t_vector incident, t_vector normal)
{
	float		dot_coeff;
	t_vector	reflected;

	dot_coeff = dot_prod(scale_vector(incident, 2), normal);
	reflected = scale_vector(normal, -dot_coeff);
	reflected.x += incident.x;
	reflected.y += incident.y;
	reflected.z += incident.z;
	return (normalize_vec(reflected));
}

__device__ t_vector		refracted_vector(t_object ray, t_object object, float next_refraction_index)
{
	t_vector	normal;
	t_vector	refracted;
	float		incident_cos;
	float		refraction_indexes_ratio;

	normal = shape_norml(ray, object);
	refraction_indexes_ratio = ray.refraction / next_refraction_index;
	if (fabs(refraction_indexes_ratio - 1) < EPSILON)
		return (ray.direction);
	incident_cos = 1.0 - pow(refraction_indexes_ratio, 2) * (1.0 - pow(dot_prod(normal, ray.direction), 2));
	refracted = scale_vector(ray.direction, refraction_indexes_ratio);
	refracted = sum_vectors(refracted, scale_vector(normal, -1 * refraction_indexes_ratio * dot_prod(ray.direction, normal)));
	refracted = sum_vectors(refracted, scale_vector(normal, -1 * sqrt(incident_cos)));
	return (normalize_vec(refracted));
}
