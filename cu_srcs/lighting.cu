#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_color			ambiant_color(t_scene scene, t_object object)
{
	t_color		ambiant_color;
	float		factor;

	factor = scene.power * object.diffuse;
	ambiant_color.r = factor * (object.color.r * scene.theme.r / 255.0);
	ambiant_color.g = factor * (object.color.g * scene.theme.g / 255.0);
	ambiant_color.b = factor * (object.color.b * scene.theme.b / 255.0);
	ambiant_color.a = 0;
	return (ambiant_color);
}

__device__ int				hiit_test(t_object clt_obj, t_object obj, t_object l_ray, float norm)
{
	if (!(l_ray.intersect && l_ray.norm > EPSILON))
		return (0);
	if (clt_obj.index == obj.index)
		return (l_ray.norm < norm - 0.1);
	return (l_ray.norm < norm);
}

__device__ t_color			filter_light_through_object(t_color initial_color, t_object object)
{
	t_color		final_color;
	float		transparency;

	transparency = fmax(object.transparency, (float)((float)object.color.a / 255.0));
	final_color = fade_color(initial_color, transparency);
	final_color.r = final_color.r * object.color.r / 255.0;
	final_color.g = final_color.g * object.color.g / 255.0;
	final_color.b = final_color.b * object.color.b / 255.0;
	return (final_color);
}

__device__ t_object		light_ray_from_shadow_ray(t_object shadow_ray, t_light light)
{
	t_object	light_ray;

	light_ray = shadow_ray;
	light_ray.direction = scale_vector(shadow_ray.direction, -1);
	light_ray.origin = (light.typpe == AMBIANT) ? shadow_ray.origin : light.posiition;
	return (light_ray);
}

__device__ t_color			diffuse_light_for_intersection(t_object light_ray, t_object ray, t_object object, t_light light)
{
	if (light.typpe == AMBIANT)
		return (ambiant_light_for_intersection(light_ray, ray, object, light));
	else if (light.typpe == PROJECTOR)
		return (projector_light_for_intersection(light_ray, ray, object, light));

	return (omni_light_for_intersection(light_ray, ray, object, light));
}
