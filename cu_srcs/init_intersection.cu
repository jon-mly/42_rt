#include "hip/hip_runtime.h"
extern "C" {
#include <rt.h>
#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_object		init_primary_ray(int x, int y, t_camera camera, float aliasing_variation, int horizontal)
{
	t_object		ray;
	t_point			projector_point;
	float			virtual_x;
	float			virtual_y;			

	if (aliasing_variation > 0 && aliasing_variation < 1)
	{
		virtual_x = (float)x + ((horizontal) ? aliasing_variation : 0);
		virtual_y = (float)y + ((!horizontal) ? aliasing_variation : 0);
	}
	else
	{
		virtual_x = (float)x + aliasing_variation;
		virtual_y = (float)y + aliasing_variation;
	}
	projector_point.x = camera.up_left_corner.x + virtual_x * camera.horizontal_vect.x + virtual_y * camera.vertical_vect.x;
	projector_point.y = camera.up_left_corner.y + virtual_x * camera.horizontal_vect.y + virtual_y * camera.vertical_vect.y;
	projector_point.z = camera.up_left_corner.z + virtual_x * camera.horizontal_vect.z + virtual_y * camera.vertical_vect.z;
	ray.direction = vector_pts(camera.spot, projector_point);
	ray.direction = normalize_vec(ray.direction);
	ray.origin = camera.spot;
	ray.refraction = 1;
	ray.reflection = 1;
	ray.transparency = 1;
	ray.intersect = FALSE;
	return (ray);
}

__device__ t_color			primary_ray(t_scene *scene, t_object *obj, t_light *light, t_object ray, int depth, int refraction)
{
	int					object_index;
	int 				closest_object_index;
	float				closest_distance;
	t_color				colorout;
	t_color				refracted_color;
	t_color				reflected_color;
	t_object			intersected_object;

	colorout = BLACK;
	reflected_color = BLACK;
	refracted_color = BLACK;
	closest_object_index = -1;
	object_index = -1;
	while (++object_index < scene->objects_count)
	{
		ray = intersect_obj(ray, obj[object_index]);
		if (ray.intersect && ((closest_object_index != -1 && ray.norm < closest_distance) || closest_object_index == -1) && ray.norm > EPSILON)
		{
			closest_object_index = object_index;
			closest_distance = ray.norm;
		}
	}
	if (closest_object_index != -1)
	{
		ray.norm = closest_distance;
		ray.intersectiion = point_from_vec(ray.origin, ray.direction, closest_distance);
		intersected_object = object_with_local_parameters(obj[closest_object_index], textured_color_if_needed(obj[closest_object_index], ray.intersectiion));
		colorout = get_color_on_inters(ray, intersected_object.index, intersected_object, scene, light, obj);
		if (intersected_object.reflection > 0)
			reflected_color = main_reflected_raytracing(scene, obj, light,
					init_reflected_ray(ray, intersected_object));
		if (intersected_object.transparency > 0)
			refracted_color = main_refracted_raytracing(scene, obj, light,
					init_refracted_ray(ray, intersected_object,
						intersected_object.refraction, intersected_object.transparency));
		colorout = add_color(colorout, add_color(refracted_color, reflected_color));
	}
	colorout = add_color(colorout, direct_light_raytracing(scene, obj, light, ray));
	return (colorout);
}
