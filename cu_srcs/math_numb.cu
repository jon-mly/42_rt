#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ float		vec_norm(t_vector vec)
{
	float		norm;

	norm = sqrt((float)(pow((float)vec.x, (float)2) + pow((float)vec.y, (float)2) + pow((float)vec.z, (float)2)));
	return (norm);
}

__device__ float		dot_prod(t_vector vect_1, t_vector vect_2)
{
	float		product;

	product = (vect_1.x * vect_2.x) +
				(vect_1.y * vect_2.y) +
				(vect_1.z * vect_2.z);
	return (product);
}

__device__ t_vector		cross_prod(t_vector vect_1, t_vector vect_2)
{
	t_vector	cross;

	cross.x = vect_1.y * vect_2.z - vect_1.z * vect_2.y;
	cross.y = vect_1.z * vect_2.x - vect_1.x * vect_2.z;
	cross.z = vect_1.x * vect_2.y - vect_1.y * vect_2.x;
	return (cross);
}

__device__ float		linear_interpolation(float a, float b, float ratio)
{
	return (a + (b - a) * ratio);
}
