#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_object		finite_cylinder_intersection(t_object ray, t_object cylinder)
{
	t_vector	distance;
	t_vector	ray_dir;
	float		a;
	float		b;
	float		c;
	float		closest_norm;
	float		farest_norm;

	distance = vector_pts(cylinder.point, ray.origin);
	ray_dir = rotate_vec_angles(cylinder, ray.direction, 0);
	distance = rotate_vec_angles(cylinder, distance, 0);
	a = powf((float)ray_dir.x, (float)2) + powf((float)ray_dir.y, (float)2);
	b = 2 * (distance.x * ray_dir.x + distance.y * ray_dir.y);
	c = powf((float)distance.x, (float)2) + powf((float)distance.y, (float)2) - powf((float)cylinder.radius, (float)2);
	closest_norm = closest_dist_quad(a, b, c);
	farest_norm = farest_distance_quadratic(a, b, c);
	ray.intersect = FALSE;
	if (closest_norm > EPSILON)
	{
		ray.norm = closest_norm;
		ray.intersectiion = point_from_vec(ray.origin, ray.direction, ray.norm);
		distance = vector_pts(cylinder.point, ray.intersectiion);
		distance = rotate_vec_angles(cylinder, distance, 0);
		ray.intersect = (distance.z >= 0 && distance.z <= cylinder.height);
	}
	if (!ray.intersect && farest_norm > EPSILON)
	{
		ray.norm = farest_norm;
		ray.intersectiion = point_from_vec(ray.origin, ray.direction, ray.norm);
		distance = vector_pts(cylinder.point, ray.intersectiion);
		distance = rotate_vec_angles(cylinder, distance, 0);
		ray.intersect = (distance.z >= 0 && distance.z <= cylinder.height);		
	}
	return (ray);
}

__device__ t_object		finite_cone_intersection(t_object ray, t_object cone)
{
	t_vector	distance;
	t_vector	ray_dir;
	float		a;
	float		b;
	float		c;
	float		k;
	float		closest_norm;
	float		farest_norm;

	distance = vector_pts(cone.center, ray.origin);
	ray_dir = rotate_vec_angles(cone, ray.direction, 0);
	distance = rotate_vec_angles(cone, distance, 0);
	k = -1 - powf((float)(tanf((float)(cone.angle))), (float)2);
	a = powf((float)vec_norm(ray_dir), (float)2) + k * powf((float)ray_dir.z, (float)2);
	b = 2 * (dot_prod(distance, ray_dir) + k * ray_dir.z * distance.z);
	c = powf((float)vec_norm(distance), (float)2) + k * powf((float)distance.z, (float)2);
	ray.intersect = FALSE;
	closest_norm = closest_dist_quad(a, b, c);
	farest_norm = farest_distance_quadratic(a, b, c);
	if (closest_norm > EPSILON)
	{
		ray.norm = closest_norm;
		ray.intersectiion = point_from_vec(ray.origin, ray.direction, ray.norm);
		distance = vector_pts(cone.center, ray.intersectiion);
		distance = rotate_vec_angles(cone, distance, 0);
		ray.intersect = (distance.z >= 0 && distance.z <= cone.height);
	}
	if (!ray.intersect && farest_norm > EPSILON)
	{
		ray.norm = farest_norm;
		ray.intersectiion = point_from_vec(ray.origin, ray.direction, ray.norm);
		distance = vector_pts(cone.center, ray.intersectiion);
		distance = rotate_vec_angles(cone, distance, 0);
		ray.intersect = (distance.z >= 0 && distance.z <= cone.height);		
	}
	return (ray);
}

__device__	t_object		cone_intersect(t_object ray, t_object cone)
{
	t_vector	distance;
	t_vector	ray_dir;
	float		a;
	float		b;
	float		c;
	float		k;

	distance = vector_pts(cone.center, ray.origin);
	ray_dir = rotate_vec_angles(cone, ray.direction, 0);
	distance = rotate_vec_angles(cone, distance, 0);
	k = -1 - powf((float)(tanf((float)(cone.angle))), (float)2);
	a = powf((float)vec_norm(ray_dir), (float)2) + k * powf((float)ray_dir.z, (float)2);
	b = 2 * (dot_prod(distance, ray_dir) + k * ray_dir.z * distance.z);
	c = powf((float)vec_norm(distance), (float)2) + k * powf((float)distance.z, (float)2);
	ray.norm = closest_dist_quad(a, b, c);
	if (ray.norm < EPSILON)
		ray.norm = farest_distance_quadratic(a, b, c);
	ray.intersect = ray.norm > 0;
	return (ray);
}

__device__ t_object			hyperboloid_intersection(t_object ray, t_object hyperboloid)
{
	t_vector		distance;
	t_vector		ray_dir;
	float			a;
	float			b;
	float			c;

	distance = vector_pts(hyperboloid.center, ray.origin);
	ray_dir = rotate_vec_angles(hyperboloid, ray.direction, 0);
	distance = rotate_vec_angles(hyperboloid, distance, 0);
	a = powf(1 / hyperboloid.rho, 2) * (powf(ray_dir.x, 2) + powf(ray_dir.y, 2))
		- powf(1 / hyperboloid.sigma, 2) * powf(ray_dir.z, 2);
	b = 2 * (powf(1 / hyperboloid.rho, 2) * (distance.x * ray_dir.x + distance.y * ray_dir.y)
		- powf(1 / hyperboloid.sigma, 2) * distance.z * ray_dir.z);
	c = powf(1 / hyperboloid.rho, 2) * (powf(distance.x, 2) + powf(distance.y, 2))
		- powf(1 / hyperboloid.sigma, 2) * powf(distance.z, 2) - 1;
	ray.norm = closest_dist_quad(a, b, c);
	ray.intersect = ray.norm > 0;
	return (ray);
}
