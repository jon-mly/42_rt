#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_object		object_with_local_parameters(t_object object, t_color local_color)
{
	t_object	local_object;

	local_object = object;
	local_object.color = local_color;
	local_object.transparency = fmaxf(object.transparency, (float)(local_color.a / 255.0));
	local_object.reflection = object.reflection;
	return (local_object);
}

__device__ t_color			textured_color_if_needed(t_object object, t_point intersection)
{
	if (object.texture_algo == PROCEDURAL)
		return (procedural_color(object, intersection));
	else if (object.texture_algo == PERLIN_ALGO)
		return (perlin_algo_texture_color(object, intersection));
	return (object.color);
}

__device__ t_color			procedural_color(t_object object, t_point intersection)
{
	if (object.texture_type == CHECKER)
		return (checker_texture_color(object, intersection));
	else if (object.texture_type == HORIZONTAL_CIRCLE || object.texture_type == VERTICAL_CIRCLE)
		return (circles_color(object, intersection, object.texture_type == VERTICAL_CIRCLE));
	else if (object.texture_type == DOTS || object.texture_type == DOTS_CROWN
		|| object.texture_type == DOTS_REVERTED || object.texture_type == DOTS_REVERTED_CROWN)
		return (dots_color(object, intersection,
			object.texture_type == DOTS_CROWN || object.texture_type == DOTS_REVERTED_CROWN,
			object.texture_type == DOTS_REVERTED || object.texture_type == DOTS_REVERTED_CROWN));
	return (object.color);
}

__device__ t_color			perlin_algo_texture_color(t_object object, t_point intersection)
{
	if (object.texture_type == PERLIN)
		return (perlin_color(object, intersection));
	else if (object.texture_type == WOOD)
		return (wood_color(object, intersection));
	else if (object.texture_type == MARBLE)
		return (marble_color(object, intersection));
	return (TRANSPARENT);
}
