#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_color			checker_texture_color(t_object object, t_point intersection)
{
	t_color		alternate;
	t_point		adjusted;
	int			x_even;
	int			y_even;
	int			z_even;

	alternate = TRANSPARENT;
	adjusted = scale_vector(intersection, 1.0 / CHECKER_WIDTH);
	x_even = is_texture_even((int)(adjusted.x < -EPSILON) ? adjusted.x - 1 : adjusted.x);
	y_even = is_texture_even((int)(adjusted.y < -EPSILON) ? adjusted.y - 1 : adjusted.y);
	z_even = is_texture_even((int)(adjusted.z < -EPSILON) ? adjusted.z - 1 : adjusted.z);
	if (z_even)
	{
		if ((x_even && y_even)
			|| (!x_even && !y_even))
			return (object.color);
		return (alternate);
	}
	else
	{
		if ((x_even && y_even)
			|| (!x_even && !y_even))
			return (alternate);
		return (object.color);
	}
}

__device__ t_color			circles_color(t_object object, t_point intersection, int horizontal)
{
	t_color		alternate;
	t_point		adjusted;
	int			distance;

	alternate = TRANSPARENT;
	adjusted = scale_vector(intersection, 1.0 / CIRCLES_WIDTH);
	if (horizontal)
		distance = (int)sqrt((adjusted.x * adjusted.x + adjusted.z * adjusted.z));
	else
		distance = (int)sqrt((adjusted.x * adjusted.x + adjusted.y * adjusted.y));
	if (distance % 2 == 0)
		return (object.color);
	return (alternate);
}

__device__ t_color			dots_color(t_object object, t_point intersection, int invert_gradient, int reverse)
{
	float		norm;
	t_color		internal_color;
	t_color		external_color;
	t_point		closest_dot;

	closest_dot = (t_point){round(intersection.x / DOTS_SPREAD) * DOTS_SPREAD,
		round(intersection.y / DOTS_SPREAD) * DOTS_SPREAD,
		round(intersection.z / DOTS_SPREAD) * DOTS_SPREAD};
	if (reverse)
	{
		external_color = object.color;
		internal_color = TRANSPARENT;
	}
	else
	{
		internal_color = object.color;
		external_color = TRANSPARENT;
	}
	norm = pts_norm(intersection, closest_dot);
	if (norm <= DOTS_WIDTH / 2)
		return (internal_color);
	else if (norm <= DOTS_WIDTH)
	{
		if (invert_gradient)
			return (interpolate_color(internal_color, external_color,
				1 - ((norm - DOTS_WIDTH / 2) / (DOTS_WIDTH / 2))));
		else
			return (interpolate_color(internal_color, external_color,
				(norm - DOTS_WIDTH / 2) / (DOTS_WIDTH / 2)));
	}
	return (external_color);
}
