extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ int			integer_part(float value)
{
	// return ((int)value);
	if (value >= 0)
		return ((int)value);
	return ((int)value - 1);
	// return ((int)value - (value < 0));
}

__device__ float		fractional_part(float value)
{
	float		integer_part;

	// integer_part = (float)((int)value);
	if (value >= 0)
		integer_part = (float)((int)value);
	else
		integer_part = (float)((int)value - 1);
	// integer_part = (float)((int)value - (value < 0));
	return (value - integer_part);
}
