#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_color			get_color_on_inters(t_object ray, int closest_object_index, t_object intersected_object, t_scene *scene, t_light *light, t_object *obj)
{
	t_object	light_ray;
	t_object	shadow_ray;
	int			light_index;
	int			object_index;
	float		norm;
	t_color		coloration;
	t_object	object_inbetween;
	int 		light_goes_through;

	light_index = -1;
	coloration = ambiant_color(*scene, intersected_object);
	while (++light_index < scene->lights_count)
	{
		light_goes_through = 1;
		shadow_ray = get_shadow_ray(light[light_index], ray, intersected_object);
		norm = shadow_ray.norm;
		object_index = -1;
		while (++object_index < scene->objects_count && light_goes_through)
		{
			shadow_ray = intersect_obj(shadow_ray, obj[object_index]);
			if (shadow_ray.intersect && ((light[light_index].typpe == AMBIANT && shadow_ray.norm > EPSILON)
				|| (light[light_index].typpe != AMBIANT && hiit_test(intersected_object, obj[object_index], shadow_ray, norm))))
			{
				shadow_ray.intersectiion = point_from_vec(shadow_ray.origin, shadow_ray.direction, shadow_ray.norm);
				object_inbetween = object_with_local_parameters(obj[object_index], textured_color_if_needed(obj[object_index], shadow_ray.intersectiion));
				shadow_ray.color = filter_light_through_object(shadow_ray.color, object_inbetween);
			}
			light_goes_through = (!(colors_are_equals(shadow_ray.color, BLACK)));
		}
		if (light_goes_through)
		{
			light_ray = light_ray_from_shadow_ray(shadow_ray, light[light_index]);
			coloration = add_color(coloration, diffuse_light_for_intersection(light_ray, ray, intersected_object, light[light_index]));
			coloration = add_color(coloration, specular_light_for_intersection(light_ray, ray, intersected_object, light[light_index]));
		}
	}
	return (coloration);
}

__device__ t_object		get_shadow_ray(t_light light, t_object ray, t_object object)
{
	t_object		shadow_ray;
	t_vector		direction;

	if (light.typpe == AMBIANT)
		shadow_ray.direction = scale_vector(light.direction, -1);
	else
	{
		direction = vector_pts(ray.intersectiion, light.posiition);
		shadow_ray.norm = vec_norm(direction);
		shadow_ray.direction = normalize_vec(direction);
	}
	shadow_ray.origin = point_from_vec(ray.intersectiion, shadow_ray.direction, EPSILON);
	shadow_ray.intersect = FALSE;
	shadow_ray.color = light.color;
	return (shadow_ray);
}
