#include "rt.h"
#include "cu_rt.h"

extern "C" void	init_gpu(t_env *e, t_cuda gpu)
{
	gpu->img_s = WIDTH * HEIGHT;
	gpu->err = 0;
	gpu->data_img = NULL;
	gpu->render_img = NULL;
	gpu->data_obj = NULL;
	gpu->data_scene = NULL;
	gpu->data_cam = NULL;
	gpu->data_light = NULL;
	gpu->img_s = sizeof(int) * WIDTH * HEIGHT;
	gpu->size_obj = sizeof(t_object) * e->scene.objects_count;
	gpu->size_scene = sizeof(t_scene);
	gpu->size_cam = sizeof(t_camera);
	gpu->size_light = sizeof(t_light) * e->scene.lights_count;
}

extern "C" void	malloc_gpu(t_cuda gpu)
{
	if (!(gpu->render_img = (int *)malloc(gpu->img_s)))
	{
		ft_putendl("Error malloc render_img");
		exit(EXIT_FAILURE);
	}
	gpu->err = hipMalloc( (void **)&gpu->data_img, gpu->img_s);
	if (gpu->err != hipSuccess)
	{
		ft_putendl("Error hipMalloc data_img");
		exit(EXIT_FAILURE);
	}
	gpu->err = hipMalloc( (void **)&gpu->data_obj, gpu->size_obj);
	if (gpu->err != hipSuccess)
	{
		ft_putendl("Error hipMalloc gpu_obj");
		exit(EXIT_FAILURE);
	}
	gpu->err = hipMalloc( (void **)&gpu->data_scene, gpu->size_scene);
	if (gpu->err != hipSuccess)
	{
		ft_putendl("Error hipMalloc gpu_scene");
		exit(EXIT_FAILURE);
	}
	gpu->err = hipMalloc( (void **)&gpu->data_cam, gpu->size_cam);
	if (gpu->err != hipSuccess)
	{
		ft_putendl("Error hipMalloc gpu_cam");
		exit(EXIT_FAILURE);
	}
	gpu->err = hipMalloc( (void **)&gpu->data_light, gpu->size_light);
	if (gpu->err != hipSuccess)
	{
		ft_putendl("Error hipMalloc gpu_light");
		exit(EXIT_FAILURE);
	}
}

extern "C" void	copy_gpu(t_env *e, t_cuda gpu)
{
	gpu->err = hipMemcpy(gpu->data_obj, e->object, gpu->size_obj, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy host to device data_obj");
		exit(EXIT_FAILURE);
	}
	gpu->err = hipMemcpy(gpu->data_scene, e->scene, gpu->size_scene, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy host to device data_scene");
		exit(EXIT_FAILURE);
	}
	gpu->err = hipMemcpy(gpu->data_cam, e->cam, gpu->size_cam, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy host to device data_cam");
		exit(EXIT_FAILURE);
	}
	gpu->err = hipMemcpy(gpu->data_light, e->light, gpu->size_light, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy host to device data_light");
		exit(EXIT_FAILURE);
	}
}

extern "C" void	call_kernel(t_env *e, t_cuda gpu)
{
	static dim3 grid;
	static dim3 block;

	block(1024, 1024);
	grid(WIDTH/block.x, WIDTH/block.y);
	pix_raytracing <<< grid, block >>> (gpu->render_img, gpu->data_scene, gpu->data_cam, gpu->data_obj, gpu->data_light);
	gpu->err = hipGetLastError();
	if (gpu->err != hipSuccess)
	{
		ft_putendl("Error kernel");
		exit(EXIT_FAILURE);
	}
	gpu->err = hipMemcpy(gpu->render_img, gpu->data_img, gpu->img_s, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy host to device data_light");
		exit(EXIT_FAILURE);
	}
}
