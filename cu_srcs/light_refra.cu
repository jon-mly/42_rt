#include "hip/hip_runtime.h"
extern "C" {
#include <rt.h>
#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_object		init_refracted_ray(t_object original_ray, t_object intersected_object, float next_refraction, float next_transparency)
{
	t_object		ray;

	ray.direction = refracted_vector(original_ray, intersected_object, next_refraction);
	ray.origin = point_from_vec(original_ray.intersectiion, ray.direction, EPSILON);
	ray.intersect = FALSE;
	ray.transparency = next_transparency;
	ray.refraction = next_refraction;
	ray.reflection = intersected_object.reflection * original_ray.reflection;
	return (ray);
}

__device__ t_color			main_refracted_raytracing(t_scene *scene, t_object *obj, t_light *light, t_object ray)
{
	int					object_index;
	int 				closest_object_index;
	float				closest_distance;
	t_color				added_color;
	t_color				colorout;
	int					iter_count;
	int					max_iterations;
	int					inside_object;
	int					current_object_id;
	t_object			intersected_object;

	iter_count = -1;
	max_iterations = MAX_DEPTH;
	colorout = BLACK;
	current_object_id = -1;
	inside_object = FALSE;
	while (++iter_count < max_iterations * 2) {
		added_color = BLACK;
		closest_object_index = -1;
		object_index = -1;
		while (++object_index < scene->objects_count)
		{
			ray = intersect_obj(ray, obj[object_index]);
			if (ray.intersect && ((closest_object_index != -1 && ray.norm < closest_distance)
						|| closest_object_index == -1) && ray.norm > EPSILON)
			{
				closest_object_index = object_index;
				closest_distance = ray.norm;
			}
		}
		if (closest_object_index != -1)
		{
			if (inside_object && current_object_id == obj[closest_object_index].id)
				inside_object = FALSE;
			else if (!inside_object)
			{
				current_object_id = obj[closest_object_index].id;
				inside_object = TRUE;
			}
			ray.norm = closest_distance;
			ray.intersectiion = point_from_vec(ray.origin, ray.direction, closest_distance);
			intersected_object = object_with_local_parameters(obj[closest_object_index],
					textured_color_if_needed(obj[closest_object_index], ray.intersectiion));
			added_color = get_color_on_inters(ray, intersected_object.index, intersected_object, scene, light, obj);
			added_color = fade_color(added_color, ray.transparency);
			if (intersected_object.reflection > 0)
				added_color = add_color(added_color, ponctual_reflected_raytracing(scene, obj, light,
							init_reflected_ray(ray, intersected_object)));
			colorout = add_color(colorout, added_color);
		}
		if (closest_object_index == -1 || intersected_object.transparency == 0)
			return (colorout);
		if (!inside_object)
			ray = init_refracted_ray(ray, intersected_object,
					1, ray.transparency);
		else
			ray = init_refracted_ray(ray, intersected_object,
					intersected_object.refraction, intersected_object.transparency);


	}
	return (colorout);
}

__device__ t_color			ponctual_refracted_raytracing(t_scene *scene, t_object *obj, t_light *light, t_object ray)
{
	int					object_index;
	int 				closest_object_index;
	float				closest_distance;
	t_color				added_color;
	t_color				colorout;
	int					iter_count;
	int					max_iterations;
	int					inside_object;
	int					current_object_id;
	t_object			intersected_object;

	iter_count = -1;
	max_iterations = MAX_DEPTH;
	colorout = BLACK;
	current_object_id = -1;
	inside_object = FALSE;
	while (++iter_count < max_iterations * 2) {
		added_color = BLACK;
		closest_object_index = -1;
		object_index = -1;
		while (++object_index < scene->objects_count)
		{
			ray = intersect_obj(ray, obj[object_index]);
			if (ray.intersect && ((closest_object_index != -1 && ray.norm < closest_distance)
						|| closest_object_index == -1) && ray.norm > EPSILON)
			{
				closest_object_index = object_index;
				closest_distance = ray.norm;
			}
		}
		if (closest_object_index != -1)
		{
			if (inside_object && current_object_id == obj[closest_object_index].id)
				inside_object = FALSE;
			else if (!inside_object)
			{
				current_object_id = obj[closest_object_index].id;
				inside_object = TRUE;
			}
			ray.norm = closest_distance;
			ray.intersectiion = point_from_vec(ray.origin, ray.direction, closest_distance);
			intersected_object = object_with_local_parameters(obj[closest_object_index],
					textured_color_if_needed(obj[closest_object_index], ray.intersectiion));
			added_color = get_color_on_inters(ray, intersected_object.index, intersected_object, scene, light, obj);
			added_color = fade_color(added_color, ray.transparency);
			colorout = add_color(colorout, added_color);
		}
		if (closest_object_index == -1 || intersected_object.transparency == 0)
			return (colorout);
		if (!inside_object)
			ray = init_refracted_ray(ray, intersected_object,
					1, ray.transparency);
		else
			ray = init_refracted_ray(ray, intersected_object,
					intersected_object.refraction, intersected_object.transparency);


	}
	return (colorout);
}
