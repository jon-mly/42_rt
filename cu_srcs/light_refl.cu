#include "hip/hip_runtime.h"
extern "C" {
#include <rt.h>
#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_object		init_reflected_ray(t_object original_ray, t_object intersected_object)
{
	t_object	ray;

	ray.direction = reflected_vector(original_ray.direction, shape_norml(original_ray, intersected_object));
	ray.direction = normalize_vec(ray.direction);
	ray.origin = point_from_vec(original_ray.intersectiion, ray.direction, EPSILON);
	ray.intersect = FALSE;
	ray.reflection = intersected_object.reflection * original_ray.reflection * (1 - intersected_object.transparency);
	ray.refraction = intersected_object.refraction;
	ray.transparency = intersected_object.transparency;
	return (ray);
}

__device__ t_color			main_reflected_raytracing(t_scene *scene, t_object *obj, t_light *light, t_object ray)
{
	int					object_index;
	int 				closest_object_index;
	float				closest_distance;
	t_color				added_color;
	int					iter_count;
	t_color				colorout;
	int					max_iterations;
	t_object			intersected_object;

	iter_count = -1;
	max_iterations = MAX_DEPTH;
	colorout = BLACK;
	while (++iter_count < max_iterations) {
		added_color = BLACK;
		closest_object_index = -1;
		object_index = -1;
		while (++object_index < scene->objects_count)
		{
			ray = intersect_obj(ray, obj[object_index]);
			if (ray.intersect && ((closest_object_index != -1 && ray.norm < closest_distance)
						|| closest_object_index == -1) && ray.norm > EPSILON)
			{
				closest_object_index = object_index;
				closest_distance = ray.norm;
			}
		}
		if (closest_object_index != -1)
		{
			ray.norm = closest_distance;
			ray.intersectiion = point_from_vec(ray.origin, ray.direction, closest_distance);
			intersected_object = object_with_local_parameters(obj[closest_object_index],
					textured_color_if_needed(obj[closest_object_index], ray.intersectiion));
			added_color = get_color_on_inters(ray, intersected_object.index, intersected_object, scene, light, obj);
			added_color = fade_color(added_color, ray.reflection);
			if (intersected_object.transparency > 0)
				added_color = add_color(added_color, ponctual_refracted_raytracing(scene, obj, light,
							init_refracted_ray(ray, intersected_object,
								intersected_object.refraction, intersected_object.transparency)));
		}
		colorout = add_color(colorout, added_color);
		if (closest_object_index == -1 || intersected_object.reflection == 0)
			return (colorout);
		ray = init_reflected_ray(ray, intersected_object);
	}
	return (colorout);
}

__device__ t_color			ponctual_reflected_raytracing(t_scene *scene, t_object *obj, t_light *light, t_object ray)
{
	int					object_index;
	int 				closest_object_index;
	float				closest_distance;
	t_color				added_color;
	int					iter_count;
	t_color				colorout;
	int					max_iterations;
	t_object			intersected_object;

	iter_count = -1;
	max_iterations = 2;
	colorout = BLACK;
	while (++iter_count < max_iterations) {
		added_color = BLACK;
		closest_object_index = -1;
		object_index = -1;
		while (++object_index < scene->objects_count)
		{
			ray = intersect_obj(ray, obj[object_index]);
			if (ray.intersect && ((closest_object_index != -1 && ray.norm < closest_distance)
						|| closest_object_index == -1) && ray.norm > EPSILON)
			{
				closest_object_index = object_index;
				closest_distance = ray.norm;
			}
		}
		if (closest_object_index != -1)
		{
			ray.norm = closest_distance;
			ray.intersectiion = point_from_vec(ray.origin, ray.direction, closest_distance);
			intersected_object = object_with_local_parameters(obj[closest_object_index],
					textured_color_if_needed(obj[closest_object_index], ray.intersectiion));
			added_color = get_color_on_inters(ray, intersected_object.index, intersected_object, scene, light, obj);
			added_color = fade_color(added_color, ray.reflection);
		}
		colorout = add_color(colorout, added_color);
		if (closest_object_index == -1 || intersected_object.reflection == 0)
			return (colorout);
		ray = init_reflected_ray(ray, intersected_object);
	}
	return (colorout);
}
