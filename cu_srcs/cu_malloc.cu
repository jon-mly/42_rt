extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void	init_gpu(t_env *e, t_cuda *gpu)
{
	gpu->img_s = WIN_WIDTH * WIN_HEIGHT;
	gpu->d_img = NULL;
	gpu->render_img = NULL;
	gpu->d_obj = NULL;
	gpu->d_scene = NULL;
	gpu->d_cam = NULL;
	gpu->d_light = NULL;
	gpu->img_s = sizeof(int) * WIN_WIDTH * WIN_HEIGHT;
	gpu->size_obj = sizeof(t_object) * e->scene.objects_count;
	gpu->size_scene = sizeof(t_scene);
	gpu->size_cam = sizeof(t_camera);
	gpu->size_light = sizeof(t_light) * e->scene.lights_count;
}

void checkGpuMem()
{

	float free_m,total_m,used_m;

	size_t free_t,total_t;

	hipMemGetInfo(&free_t,&total_t);

	free_m =(uint)free_t/1048576.0 ;

	total_m=(uint)total_t/1048576.0;

	used_m=total_m-free_m;

	printf ( "  mem free %lu .... %f MB mem total %lu....%f MB mem used %f MB\n",free_t,free_m,total_t,total_m,used_m);

}

void	malloc_gpu(t_env *e, t_cuda *gpu)
{
	hipError_t			err;
	
	if (!(gpu->render_img = (int *)malloc(gpu->img_s)))
	{
		ft_putendl("Error malloc render_img");
		exit(EXIT_FAILURE);
	}
	checkGpuMem();
	err = hipMalloc( (void **)&gpu->d_img, gpu->img_s);
	if (err != hipSuccess)
	{
		ft_putendl("Error hipMalloc d_img");
		exit(EXIT_FAILURE);
	}
	//checkGpuMem();
	err = hipMalloc( (void **)&gpu->d_obj, gpu->size_obj);
	if (err != hipSuccess)
	{
		ft_putendl("Error hipMalloc d_obj");
		exit(EXIT_FAILURE);
	}
	err = hipMalloc( (void **)&gpu->d_scene, gpu->size_scene);
	if (err != hipSuccess)
	{
		ft_putendl("Error hipMalloc d_scene");
		exit(EXIT_FAILURE);
	}
	err = hipMalloc( (void **)&gpu->d_cam, gpu->size_cam);
	if (err != hipSuccess)
	{
		ft_putendl("Error hipMalloc d_cam");
		exit(EXIT_FAILURE);
	}
	err = hipMalloc( (void **)&gpu->d_light, gpu->size_light);
	if (err != hipSuccess)
	{
		ft_putendl("Error hipMalloc d_light");
		exit(EXIT_FAILURE);
	}
}

void	copy_gpu(t_env *e, t_cuda *gpu)
{
	hipError_t			err;
	
	err = hipMemcpy(gpu->d_obj, e->scene.objects, gpu->size_obj, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy host to device d_obj");
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(gpu->d_scene, &e->scene, gpu->size_scene, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy host to device d_scene");
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(gpu->d_cam, &e->camera, gpu->size_cam, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy host to device d_cam");
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(gpu->d_light, e->scene.lights, gpu->size_light, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy host to device d_light");
		exit(EXIT_FAILURE);
	}
}
