#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_object		sphere_intersect(t_object ray, t_object sphere)
{
	float		a;
	float		b;
	float		c;
	t_vector	distance;

	distance = vector_pts(sphere.center, ray.origin);
	a = dot_prod(ray.direction, ray.direction);
	b = 2 * dot_prod(ray.direction, distance);
	c = dot_prod(distance, distance) - pow((float)sphere.radius, (float)2);
	ray.norm = closest_dist_quad(a, b, c);
	ray.intersect = ray.norm > 0;
	return (ray);
}

__device__ t_object	plane_intersect(t_object ray, t_object plane)
{
	if (dot_prod(plane.normal, ray.direction) == 0)
	{
		ray.intersect = FALSE;
		return (ray);
	}
	ray.norm = (dot_prod(plane.normal, plane.point) - dot_prod(plane.normal,
			ray.origin)) / dot_prod(plane.normal, ray.direction);
	ray.intersect = (ray.norm > 0 || ray.norm < 10000000);
	return (ray);
}

__device__ t_object		cylinder_intersect(t_object ray, t_object cylinder)
{
	t_vector	distance;
	t_vector	ray_dir;
	float		a;
	float		b;
	float		c;

	distance = vector_pts(cylinder.point, ray.origin);
	ray_dir = rotate_vec_angles(cylinder, ray.direction, 0);
	distance = rotate_vec_angles(cylinder, distance, 0);
	a = pow((float)ray_dir.x, (float)2) + pow((float)ray_dir.y, (float)2);
	b = 2 * (distance.x * ray_dir.x + distance.y * ray_dir.y);
	c = pow((float)distance.x, (float)2) + pow((float)distance.y, (float)2) - pow((float)cylinder.radius, (float)2);
	ray.norm = closest_dist_quad(a, b, c);
	ray.intersect = ray.norm > 0;
	return (ray);
}
