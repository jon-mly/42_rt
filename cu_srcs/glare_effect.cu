#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_color			direct_light_raytracing(t_scene *scene, t_object *obj, t_light *light, t_object ray)
{
	t_color		glare_color;
	t_light		current_light;
	t_object	associated_plane;
	float		distance_from_origin;
	int			light_index;

	light_index = -1;
	glare_color = BLACK;
	while (++light_index < scene->lights_count)
	{
		current_light = light[light_index];
		if (current_light.typpe != AMBIANT)
			continue;		
		associated_plane = light_plane(ray, current_light);
		ray = intersect_obj(ray, associated_plane);
		if (ray.intersect)
		{
			distance_from_origin = pts_norm(ray.intersectiion, current_light.posiition);
			if (distance_from_origin <= current_light.power / 10)
				glare_color = add_color(glare_color,
					glare_color_from_distance(distance_from_origin, current_light));
		}
	}
	return (glare_color);
}

__device__ t_object		light_plane(t_object ray, t_light light)
{
	t_object	plane;

	// plane = (t_object){
	// 	.point = light.posiition,
	// 	.normal = scale_vector(ray.direction, -1)
	// };
	plane.typpe = PLANE;
	plane.point = light.posiition;
	plane.normal = scale_vector(ray.direction, -1);
	return (plane);
}

__device__ t_color			glare_color_from_distance(float distance, t_light light)
{
	float		intensity;

	intensity = pow((light.power / 10 - distance) / (light.power / 10), 4);
	return (fade_color(light.color, intensity));
}
