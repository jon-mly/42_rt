#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_color			raytracing(int x, int y, t_scene *scene, t_camera *camera, t_object *obj, t_light *light, float aliasing_var)
{
	t_object			ray;
	t_color				horizontal_color;
	t_color				vertical_color;

	ray = init_primary_ray(x, y, *camera, aliasing_var, 0);
	horizontal_color = primary_ray(scene, obj, light, ray, 0, 0);
	if (aliasing_var == 0 || aliasing_var == 1)
		return (horizontal_color);
	ray = init_primary_ray(x, y, *camera, aliasing_var, 1);
	vertical_color = primary_ray(scene, obj, light, ray, 0, 0);
	return (average_color(horizontal_color, vertical_color));
}

__global__ void pixRaytracing(int *img, t_scene *scene, t_camera *cam, t_object *obj, t_light *light)
{
	int		x;
	int		y;
	int		idx;
	int		aliasing_iter;
	float	aliasing_var;
	t_color	average;

	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;
	idx = y * WIN_WIDTH + x;
	if (x == 0 && y == 0)
	{
		printf("camera width: %f\n", cam->width);
		printf("camera height: %f\n", cam->height);
	}
	aliasing_iter = -1;
	while (++aliasing_iter < ALIASING)
	{
		aliasing_var = (float)aliasing_iter / (float)ALIASING;
		if (aliasing_iter == 0 || aliasing_iter == 1)
			average = raytracing(x, y, scene, cam, obj, light, aliasing_var);
		else
			average = average_color(average, raytracing(x, y, scene, cam, obj, light, aliasing_var));
	}
	if (x == 0 && y == 0)
		printf("average: %d\n", color_to_int(average));
	img[idx] = color_to_int(average);
}

void	call_kernel(t_env *e, t_cuda *gpu)
{
	hipError_t			err;
	static dim3 block(8, 8);
	static dim3 grid(WIN_WIDTH/block.x, WIN_HEIGHT/block.y);
	
	printf("camera width: %f\n", e->camera.width);
	printf("camera height: %f\n", e->camera.height);
	//printf("block.x = %d\n block.y = %d\n", block.x, block.y);
	//printf("grid.x = %d\n grid.y = %d\n", grid.x, grid.y);
	pixRaytracing<<<1, 1>>>(gpu->d_img, gpu->d_scene, gpu->d_cam, gpu->d_obj, gpu->d_light);
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		ft_putendl("Error kernel");
		printf("error: %d %s\n", err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(gpu->render_img, gpu->d_img, gpu->img_s, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		ft_putendl("Problem cpy device to host d_img");
		printf("error: %d\n", err);
		exit(EXIT_FAILURE);
	}
}
