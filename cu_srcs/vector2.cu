#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_vector		sum_vectors(t_vector vect1, t_vector vect2)
{
	t_vector	sum;

	sum.x = vect1.x + vect2.x;
	sum.y = vect1.y + vect2.y;
	sum.z = vect1.z + vect2.z;
	return (sum);
}

__device__ t_vector		scale_vector(t_vector vect, float scale)
{
	vect.x *= scale;
	vect.y *= scale;
	vect.z *= scale;
	return (vect);
}

__device__ t_vector		rotate_vec_angles(t_object reference, t_vector vect, int reverse)
{
	if (!reverse)
	{
		vect = vec_rotate_y(vect, reference.y_angle, reverse);
		vect = vec_rotate_x(vect, reference.x_angle, reverse);
	}
	else
	{
		vect = vec_rotate_x(vect, reference.x_angle, reverse);
		vect = vec_rotate_y(vect, reference.y_angle, reverse);
	}
	return (vect);
}

__device__ t_vector		vec_rotate_x(t_vector vector, float angle, int inverse)
{
	t_vector	rotated;

	if (!inverse)
	{
		rotated.x = vector.x;
		rotated.y = vector.y * cosf(angle) + vector.z * sinf(angle);
		rotated.z = -vector.y * sinf(angle) + vector.z * cosf(angle);
	}
	else
	{
		rotated.x = vector.x;
		rotated.y = cosf(angle) * vector.y - sinf(angle) * vector.z;
		rotated.z = sinf(angle) * vector.y + cosf(angle) * vector.z;
	}
	return (rotated);
}

__device__ t_vector		vec_rotate_y(t_vector vector, float angle, int inverse)
{
	t_vector	rotated;

	if (!inverse)
	{
		rotated.x = vector.x * cosf(angle) - vector.z * sinf(angle);
		rotated.y = vector.y;
		rotated.z = vector.x * sinf(angle) + vector.z * cosf(angle);
	}
	else
	{
		rotated.x = cosf(angle) * vector.x + sinf(angle) * vector.z;
		rotated.y = vector.y;
		rotated.z = -sinf(angle) * vector.x + cosf(angle) * vector.z;
	}
	return (rotated);
}
