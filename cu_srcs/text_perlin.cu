#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ t_color			perlin_color(t_object object, t_point intersection)
{
	t_point		altered_coordinates;
	t_color		color;
	float		noise;

	altered_coordinates = intersection;
	noise = perlin_noise(10, 0.01, 0.25, altered_coordinates) * 2;
	color.r = bounded_color_value((1 - noise) / 2 * 255, 0, 255);
	color.g = bounded_color_value((1 - noise) / 2 * 255, 0, 255);
	color.b = bounded_color_value((1 - noise) / 2 * 255, 0, 255);
	color.a = 0;
	// color.a = bounded_color_value((1 - noise) * 255, 0, 255);
	return (color);
}

__device__ t_color			wood_color(t_object object, t_point intersection)
{
	t_point		altered_coordinates;
	t_color		color;
	float		noise;

	altered_coordinates = intersection;
	noise = 20 * perlin_noise(5, 0.005, 0.20, altered_coordinates);
	noise -= (int)noise;
	color.r = bounded_color_value((1 - noise) / 2 * 255, 0, 255);
	color.g = bounded_color_value((1 - noise) / 2 * 255, 0, 255);
	color.b = bounded_color_value((1 - noise) / 2 * 255, 0, 255);
	color.a = 0;
	return (color);
}

__device__ t_color			marble_color(t_object object, t_point intersection)
{
	t_point		altered_coordinates;
	t_color		color;
	float		noise;

	altered_coordinates = intersection;
	noise = altered_coordinates.x / 5 + altered_coordinates.y / 18 + altered_coordinates.z / 63;
	noise = cosf(noise + perlin_noise(10, 0.1, 0.10, altered_coordinates) * 5);
	color.r = bounded_color_value((1 - noise) / 2 * 255, 0, 255);
	color.g = bounded_color_value((1 - noise) / 2 * 255, 0, 255);
	color.b = bounded_color_value((1 - noise) / 2 * 255, 0, 255);
	color.a = 0;
	return (color);
}

__device__ float			perlin_noise(int octaves, float frequency, float persistence, t_point point)
{
	float		noise;
	//float		amplitude;
	//float		geometric_limit;
	int			octave;
	t_point		values;

	octave = -1;
	//amplitude = 1.0;
	noise = 0;
	while (++octave < octaves)
	{
		values = (t_point){point.x * frequency * octave,
			point.y * frequency * octave,
			point.z * frequency * octave};
		noise += (1 / octave) * get_perlin_noise_value(values.x, values.y, values.z);
	}
	//geometric_limit = (1 - persistence) / (1 - amplitude);
	// return (noise * geometric_limit);
	// return ((noise * geometric_limit) / 2 + 0.5);
	// return ((1 + noise) / 2);
	return (noise);
}

__device__ float			get_perlin_noise_value(float x, float y, float z)
{
	int			int_x;
	int			int_y;
	int			int_z;
	float		square_noises[8];
	float		polynomials_factors[3];
	float		horizontal_interpolations[4];
	float		vertical_interpolations[2];

	int_x = integer_part(x) % 256;
	int_y = integer_part(y) % 256;
	int_z = integer_part(z) % 256;
	x = fractional_part(x);
	y = fractional_part(y);
	z = fractional_part(z);

	square_noises[0] = dot_prod(get_random_gradient(int_x, int_y, int_z), (t_vector){x, y, z});
	square_noises[1] = dot_prod(get_random_gradient(int_x, int_y, int_z + 1), (t_vector){x, y, z - 1.0});
	square_noises[2] = dot_prod(get_random_gradient(int_x, int_y + 1, int_z), (t_vector){x, y - 1.0, z});
	square_noises[3] = dot_prod(get_random_gradient(int_x, int_y + 1, int_z + 1), (t_vector){x, y - 1.0, z - 1.0});
	square_noises[4] = dot_prod(get_random_gradient(int_x + 1, int_y, int_z), (t_vector){x - 1.0, y, z});
	square_noises[5] = dot_prod(get_random_gradient(int_x + 1, int_y, int_z + 1), (t_vector){x - 1.0, y, z - 1.0});
	square_noises[6] = dot_prod(get_random_gradient(int_x + 1, int_y + 1, int_z), (t_vector){x - 1.0, y - 1.0, z});
	square_noises[7] = dot_prod(get_random_gradient(int_x + 1, int_y + 1, int_z + 1), (t_vector){x - 1.0, y - 1.0, z - 1.0});

	polynomials_factors[0] = perlin_polynom(x);
	polynomials_factors[1] = perlin_polynom(y);
	polynomials_factors[2] = perlin_polynom(z);

	horizontal_interpolations[0] = linear_interpolation(square_noises[0], square_noises[4], polynomials_factors[0]);
	horizontal_interpolations[1] = linear_interpolation(square_noises[2], square_noises[6], polynomials_factors[0]);
	horizontal_interpolations[2] = linear_interpolation(square_noises[1], square_noises[5], polynomials_factors[0]);
	horizontal_interpolations[3] = linear_interpolation(square_noises[3], square_noises[7], polynomials_factors[0]);

	vertical_interpolations[0] = linear_interpolation(horizontal_interpolations[0], horizontal_interpolations[1], polynomials_factors[1]);
	vertical_interpolations[1] = linear_interpolation(horizontal_interpolations[2], horizontal_interpolations[3], polynomials_factors[1]);

	return (linear_interpolation(vertical_interpolations[0], vertical_interpolations[1], polynomials_factors[2]));
}
