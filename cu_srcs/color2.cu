#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ float		closest_dist_quad(float a, float b, float c)
{
	float		discriminant;
	float		x1;
	float		x2;
	float 		buffer;

	discriminant = b * b - 4 * a * c;
	if (discriminant < 0)
		return (-1);
	x1 = (-b - sqrt(discriminant)) / (2 * a);
	x2 = (-b + sqrt(discriminant)) / (2 * a);
	if (x2 < x1)
	{
		buffer = x2;
		x2 = x1;
		x1 = buffer;
	}
	if (x1 <= 0 && x2 <= 0)
		return (-1);
	else if (x2 > 0 && x1 <= 0)
		return (x2);
	return (x1);
}

__device__ int			is_texture_even(int value)
{
	if (value >= 0)
		return (value % 2 == 0);
	else
		return ((value) % 2 == 0);
}

__device__ t_color		interpolate_color(t_color c1, t_color c2, float ratio)
{
	t_color		result;

	result.r = fminf(fmaxf(c1.r + (c2.r - c1.r) * ratio, 0), 255);
	result.g = fminf(fmaxf(c1.g + (c2.g - c1.g) * ratio, 0), 255);
	result.b = fminf(fmaxf(c1.b + (c2.b - c1.b) * ratio, 0), 255);
	result.a = fminf(fmaxf(c1.a + (c2.a - c1.a) * ratio, 0), 255);
	return (result);
}

__device__ float		bounded_color_value(float color_value, float min_value, float max_value)
{
	if (color_value < min_value)
		return (min_value);
	else if (color_value > max_value)
		return (max_value);
	return (color_value);
}

__device__ float		farest_distance_quadratic(float a, float b, float c)
{
	float		discriminant;
	float		x1;
	float		x2;
	float 		buffer;

	discriminant = b * b - 4 * a * c;
	if (discriminant < 0)
		return (-1);
	x1 = (-b - sqrt(discriminant)) / (2 * a);
	x2 = (-b + sqrt(discriminant)) / (2 * a);
	if (x2 < x1)
	{
		buffer = x2;
		x2 = x1;
		x1 = buffer;
	}
	if (x1 <= 0 && x2 <= 0)
		return (-1);
	else if (x2 > 0 && x1 <= 0)
		return (x2);
	return (x2);
}
