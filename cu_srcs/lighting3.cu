#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "cu_rt.h"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ int			omni_color_coord(float cosinus, float distance, int obj_color, int light_color)
{
	float	distance_factor;
	float	k;
	float	color_value;

	distance_factor = 0.02 * pow((float)(distance / 1.3), (float)2) + 1;
	k = sqrt(-cosinus) / distance_factor;
	color_value = ((float)obj_color + (float)light_color) * k;
	color_value = maximize_color_value((int)color_value);
	return ((int)color_value);
}
